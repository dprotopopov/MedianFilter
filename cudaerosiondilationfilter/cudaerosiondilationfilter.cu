﻿char *title = "Erosion and Dilation filter";
char *description = "Erosion and Dilation filter";
/*
Фильтр «минимум» – также известный как фильтр эрозии, заменяет значение минимальным в окрестности.
Фильтр «максимум» – также известный как фильтр расширения, заменяет значение максимальным в окрестности.
*/

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>


#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#ifndef uint8_t
typedef unsigned char uint8_t;
#endif

#ifndef uint16_t
typedef unsigned short uint16_t;
#endif

#ifndef uint32_t
typedef unsigned int uint32_t;
#endif

// http://en.wikipedia.org/wiki/BMP_file_format
#define BMPMAGIC	0x00
#define BMPFILESIZE	0x02
#define BMPOFFSET	0x0A
#define BMPDIBSISE	0x0E
#define BMPWIDTH	0x12
#define BMPHEIGHT	0x16
#define BMPBITSPERPIXEL	0x1C

typedef struct {
	uint8_t magic[2];   /* the magic number used to identify the BMP file:
						0x42 0x4D (Hex code points for B and M).
						The following entries are possible:
						BM - Windows 3.1x, 95, NT, ... etc
						BA - OS/2 Bitmap Array
						CI - OS/2 Color Icon
						CP - OS/2 Color Pointer
						IC - OS/2 Icon
						PT - OS/2 Pointer. */
	uint32_t filesz;    /* the size of the BMP file in bytes */
	uint16_t creator1;  /* reserved. */
	uint16_t creator2;  /* reserved. */
	uint32_t offset;    /* the offset, i.e. starting address,
						of the byte where the bitmap data can be found. */
} bmp_header_t;

typedef struct {
	uint32_t header_sz;     /* the size of this header (40 bytes) */
	uint32_t width;         /* the bitmap width in pixels */
	uint32_t height;        /* the bitmap height in pixels */
	uint16_t nplanes;       /* the number of color planes being used.
							Must be set to 1. */
	uint16_t depth;         /* the number of bits per pixel,
							which is the color depth of the image.
							Typical values are 1, 4, 8, 16, 24 and 32. */
	uint32_t compress_type; /* the compression method being used.
							See also bmp_compression_method_t. */
	uint32_t bmp_bytesz;    /* the image size. This is the size of the raw bitmap
							data (see below), and should not be confused
							with the file size. */
	uint32_t hres;          /* the horizontal resolution of the image.
							(pixel per meter) */
	uint32_t vres;          /* the vertical resolution of the image.
							(pixel per meter) */
	uint32_t ncolors;       /* the number of colors in the color palette,
							or 0 to default to 2<sup><i>n</i></sup>. */
	uint32_t nimpcolors;    /* the number of important colors used,
							or 0 when every color is important;
							generally ignored. */
} bmp_dib_v3_header_t;


__global__ void global_set2grayscale(
	float *grayscale,
	unsigned int *input,
	float weight,
	int width, 
	int height)
{
	for (int id = blockDim.x*blockIdx.x + threadIdx.x;
		id < width*height;
		id += blockDim.x*gridDim.x) {
			grayscale[id] += weight*(float)(input[id]&0xFF);
	}
}

__global__ void global_add2grayscale(
	float *grayscale,
	unsigned int *input,
	float weight,
	int width, 
	int height)
{
	for (int id = blockDim.x*blockIdx.x + threadIdx.x;
		id < width*height;
		id += blockDim.x*gridDim.x) {
			grayscale[id] += weight*(float)(input[id]&0xFF);
	}
}

// Поиск индекса элемента с наилучшим значением интенсивности
__global__ void global_find2grayscale(
	float *grayscale,
	unsigned int *index,
	int width, 
	int height, 
	float radius2, 
	int mode)
{
	int radius = 1;
	while((float)radius*radius<radius2) radius++; // поскольку числа маленькие, то быстрее перебором
	if(mode>0){ // если расширение
		for (int id = blockDim.x*blockIdx.x + threadIdx.x;
			id < width*height;
			id += blockDim.x*gridDim.x) {
				int x=id%width;
				int y=id/width;
				int currentId = id;
				// перебираем пиксели в окружности
				for(int i=max(0,x-radius);i<=min(width,x+radius);i++){
					for(int j=max(0,y-radius);j<=min(height,y+radius);j++){
						if((float)((i-x)*(i-x)+(j-y)*(j-y))<=radius2) {
							int itemId = j*width+i;
							if(grayscale[itemId]>grayscale[currentId]) currentId=itemId;
						}
					}
				}
				index[id] = currentId;
		}
	} else if(mode<0){ // если эрозия
		for (int id = blockDim.x*blockIdx.x + threadIdx.x;
			id < width*height;
			id += blockDim.x*gridDim.x) {
				int x=id%width;
				int y=id/width;
				int currentId = id;
				// перебираем пиксели в окружности
				for(int i=max(0,x-radius);i<=min(width,x+radius);i++){
					for(int j=max(0,y-radius);j<=min(height,y+radius);j++){
						if((float)((i-x)*(i-x)+(j-y)*(j-y))<=radius2) {
							int itemId = j*width+i;
							if(grayscale[itemId]<grayscale[currentId]) currentId=itemId;
						}
					}
				}
				index[id] = currentId;
		}
	}else { 
		for (int id = blockDim.x*blockIdx.x + threadIdx.x;
			id < width*height;
			id += blockDim.x*gridDim.x) {
				index[id] = id;
		}
	}
}

// Получение цвета из таблицы по индексу
__global__ void global_indexcolor(
	unsigned int *input,
	unsigned int *output,
	unsigned int *index,
	int width, 
	int height)
{
	for (int id = blockDim.x*blockIdx.x + threadIdx.x;
		id < width*height;
		id += blockDim.x*gridDim.x) {
			output[id] = input[index[id]];
	}
}


__host__ void host_erosiondilationfilter(
	int gridSize, 
	int blockSize, 
	unsigned int *r, 
	unsigned int *g, 
	unsigned int *b, 
	int width, 
	int height, 
	float radius2, 
	int mode)
{
	// формула для конвертации RGB в интенсивность
	// http://en.wikipedia.org/wiki/Grayscale
	// Y = 0.2126 * R + 0.7152 * G + 0.0722 * B 
	unsigned int *channel[3] = { r,g,b };
	float weight[3] = {0.2126f, 0.7152f, 0.0722f};
	// channel - массив данных
	// width - ширина
	// height - высота

	hipError_t err;
	float *device_grayscale;
	unsigned int *device_input;
	unsigned int *device_output;
	unsigned int *device_index;

	err = hipMalloc((void**)&device_input, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_grayscale, width*height*sizeof(float));

	int blocks = (gridSize > 0)? gridSize : min(15, (int)sqrt(width*height));
	int threads = (blockSize > 0)? blockSize : min(15, (int)sqrt(width*height));

	// Шаг 1. Рассчитываем монохромное изображение
	hipMemcpy(device_input, channel[0], width*height*sizeof(unsigned int), hipMemcpyHostToDevice);
	global_set2grayscale <<< blocks, threads >>>(device_grayscale, device_input, weight[0], width, height);
	for(int j=1;j<3;j++){
		hipMemcpy(device_input, channel[j], width*height*sizeof(unsigned int), hipMemcpyHostToDevice);
		global_add2grayscale <<< blocks, threads >>>(device_grayscale, device_input, weight[j], width, height);
	}

	err = hipMalloc((void**)&device_index, width*height*sizeof(unsigned int));

	// Шаг 2. Находим индексы (то есть находим требуемую замену пикселей)
	global_find2grayscale <<< blocks, threads >>>(device_grayscale, device_index, width, height, radius2, mode);

	hipFree((void*)device_grayscale);
	err = hipMalloc((void**)&device_output, width*height*sizeof(unsigned int));

	// Шаг 3. Заменяем пиксели согласно ранее полученной подстановке
	for(int j=3;j-->0;){
		// Перевый раз копировать в видео память не надо, поскольку уже копировали при подсчёте монохромного изображения
		if(j<2) hipMemcpy(device_input, channel[j], width*height*sizeof(unsigned int), hipMemcpyHostToDevice);
		global_indexcolor <<< blocks, threads >>>(device_input, device_output, device_index, width, height);
		hipMemcpy(channel[j], device_output, width*height*sizeof(unsigned int), hipMemcpyDeviceToHost);
	}
	hipFree((void*)device_index);
	hipFree((void*)device_input);
	hipFree((void*)device_output);

	err = err;
}


int main(int argc, char* argv[])
{
	int i, j;

	std::cout << title << std::endl;

	// Find/set the device.
	int device_count = 0;
	hipGetDeviceCount(&device_count);
	for (i = 0; i < device_count; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		std::cout << "Running on GPU " << i << " (" << properties.name << ")" << std::endl;
	}

	if (argc < 5){
		printf("Usage :\t%s filter radius2 inputfilename.bmp ouputfilename.bmp\n", argv[0]);
		exit(-1);
	}

	// Получаем параметры - имена файлов

	char *filter = argv[1];
	float radius2 = atof(argv[2]);
	char *inputFileName = argv[3];
	char *outputFileName = argv[4];
	int gridSize = (argc>5)?atoi(argv[5]):0;
	int blockSize = (argc>6)?atoi(argv[6]):0;
	int mode = (strcmp(filter,"dilation")==0)?1:-1;


	printf("Title :\t%s\n", title);
	printf("Description :\t%s\n", description);
	printf("Filter :\t%s\n", filter);
	printf("Radius2 :\t%d\n", radius2);
	printf("Input file name :\t%s\n", inputFileName);
	printf("Output file name :\t%s\n", outputFileName);


	FILE *file = fopen(inputFileName, "rb");
	if (!file) { 
		fprintf(stderr, "Open file error (%s)\n", inputFileName); 
		fflush(stderr); 
		exit(-1); 
	}
	fseek(file, 0L, SEEK_END);
	long size = ftell(file);
	unsigned char *buffer = (unsigned char *)malloc((size_t)size);
	fseek(file, 0L, SEEK_SET);
	fread((void*)buffer, (size_t)1, (size_t)size, file);
	fclose(file);

	uint32_t width = *(uint32_t *)&buffer[BMPWIDTH];
	uint32_t height = *(uint32_t *)&buffer[BMPHEIGHT];
	uint32_t file_size = *(uint32_t *)&buffer[BMPFILESIZE];
	uint32_t offset = *(uint32_t *)&buffer[BMPOFFSET];
	uint16_t bits_per_pixel = *(uint16_t *)&buffer[BMPBITSPERPIXEL];
	uint16_t bytes_per_pixel = ((int)((bits_per_pixel+7)/8));
	uint32_t bytes_per_line = ((int)((bits_per_pixel * width+31)/32))*4; // http://en.wikipedia.org/wiki/BMP_file_format

	printf("BMP image size :\t%ld x %ld\n", width, height);
	printf("BMP file size :\t%ld\n", file_size);
	printf("BMP pixels offset :\t%ld\n", offset);
	printf("BMP bits per pixel :\t%d\n", bits_per_pixel);
	printf("BMP bytes per pixel :\t%d\n", bytes_per_pixel);
	printf("BMP bytes per line :\t%d\n", bytes_per_line);

	uint8_t *pixels =(uint8_t *)&buffer[offset];
	uint8_t *pixel;
	uint32_t x, y;

	/* выделение памяти под байтовые цветовые каналы */
	int count = width*height;
	unsigned int* channel[3];
	for (j = 0; j < 3; j++)
		channel[j] = (unsigned int*)malloc(count*sizeof(unsigned int));

	int pos = 0;
	for (y = 0; y < height; y++) {
		for (x = 0; x < width; x++) {
			pixel = &pixels[y*bytes_per_line+x*bytes_per_pixel];
			channel[0][pos] = (unsigned int)pixel[0];
			channel[1][pos] = (unsigned int)pixel[1];
			channel[2][pos] = (unsigned int)pixel[2];
			pos++;
		}
	}

	clock_t t1, t2;
	t1 = clock();

	host_erosiondilationfilter(gridSize, blockSize, 
		channel[0], channel[1], channel[2], 
		(int)width, (int)height,
		radius2, mode);

	t2 = clock();
	double seconds = ((double)(t2-t1))/CLOCKS_PER_SEC;
	printf("Execution time :\t%le\n", seconds);

	pos = 0;
	for (y = 0; y < height; y++) {
		for (x = 0; x < width; x++) {
			pixel = &pixels[y*bytes_per_line+x*bytes_per_pixel];
			pixel[0] = (uint8_t)channel[0][pos];
			pixel[1] = (uint8_t)channel[1][pos];
			pixel[2] = (uint8_t)channel[2][pos];
			pos++;
		}
	}
	/* высвобождаем массивы */
	for (j = 0; j < 3; j++)
		free(channel[j]);

	/* выводим результаты */
	file = fopen(outputFileName, "wb");
	if (!file) {
		fprintf(stderr, "Open file error (%s)\n", outputFileName); 
		fflush(stderr);
		exit(-1);
	}
	fwrite((void*)buffer, (size_t)1, (size_t)size, file);
	printf("Output file size :\t%ld\n", size);
	free(buffer);
	fclose(file);

	hipDeviceReset();

	exit(0);
}