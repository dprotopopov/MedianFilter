﻿char *title = "none filtering";
char *description = "none filtering";
/*
*/

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <time.h>
#include <hip/hip_runtime.h>


#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define assert( bool ) 

#ifndef uint8_t
typedef unsigned char uint8_t;
#endif

#ifndef uint16_t
typedef unsigned short uint16_t;
#endif

#ifndef uint32_t
typedef unsigned int uint32_t;
#endif

// http://en.wikipedia.org/wiki/BMP_file_format
#define BMPMAGIC	0x00
#define BMPFILESIZE	0x02
#define BMPOFFSET	0x0A
#define BMPDIBSISE	0x0E
#define BMPWIDTH	0x12
#define BMPHEIGHT	0x16
#define BMPBITSPERPIXEL	0x1C

typedef struct {
	uint8_t magic[2];   /* the magic number used to identify the BMP file:
						0x42 0x4D (Hex code points for B and M).
						The following entries are possible:
						BM - Windows 3.1x, 95, NT, ... etc
						BA - OS/2 Bitmap Array
						CI - OS/2 Color Icon
						CP - OS/2 Color Pointer
						IC - OS/2 Icon
						PT - OS/2 Pointer. */
	uint32_t filesz;    /* the size of the BMP file in bytes */
	uint16_t creator1;  /* reserved. */
	uint16_t creator2;  /* reserved. */
	uint32_t offset;    /* the offset, i.e. starting address,
						of the byte where the bitmap data can be found. */
} bmp_header_t;

typedef struct {
	uint32_t header_sz;     /* the size of this header (40 bytes) */
	uint32_t width;         /* the bitmap width in pixels */
	uint32_t height;        /* the bitmap height in pixels */
	uint16_t nplanes;       /* the number of color planes being used.
							Must be set to 1. */
	uint16_t depth;         /* the number of bits per pixel,
							which is the color depth of the image.
							Typical values are 1, 4, 8, 16, 24 and 32. */
	uint32_t compress_type; /* the compression method being used.
							See also bmp_compression_method_t. */
	uint32_t bmp_bytesz;    /* the image size. This is the size of the raw bitmap
							data (see below), and should not be confused
							with the file size. */
	uint32_t hres;          /* the horizontal resolution of the image.
							(pixel per meter) */
	uint32_t vres;          /* the vertical resolution of the image.
							(pixel per meter) */
	uint32_t ncolors;       /* the number of colors in the color palette,
							or 0 to default to 2<sup><i>n</i></sup>. */
	uint32_t nimpcolors;    /* the number of important colors used,
							or 0 when every color is important;
							generally ignored. */
} bmp_dib_v3_header_t;


__global__ void global_nonefilter(
	unsigned int *input,
	unsigned int *output,
	int width, 
	int height)
{
	for (int id = blockDim.x*blockIdx.x + threadIdx.x;
		id < width*height;
		id += blockDim.x*gridDim.x) {
			output[id] = input[id];
	}
}


__host__ void host_nonefilter(int gridSize, int blockSize, unsigned int *r, unsigned int *g, unsigned int *b, int width, int height)
{
	unsigned int *channel[3] = { r,g,b };

	// channel - массив данных
	// width - ширина
	// height - высота

	hipError_t err;
	unsigned int *device_input;
	unsigned int *device_output;

	err = hipMalloc((void**)&device_input, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_output, width*height*sizeof(unsigned int));

	int blocks = (gridSize > 0)? gridSize : min(15, (int)sqrt(width*height));
	int threads = (blockSize > 0)? blockSize : min(15, (int)sqrt(width*height));

	for(int j=0; j<3; j++) {
		hipMemcpy(device_input, channel[j], width*height*sizeof(unsigned int), hipMemcpyHostToDevice);

		global_nonefilter <<< blocks, threads >>>(device_input, device_output, width, height);

		hipMemcpy(channel[j], device_output, width*height*sizeof(unsigned int), hipMemcpyDeviceToHost);
	}

	// Освобождаем память на устройстве
	hipFree((void*)device_input);
	hipFree((void*)device_output);

	err = err;
}


int main(int argc, char* argv[])
{
	int i, j;

	std::cout << title << std::endl;

	// Find/set the device.
	int device_count = 0;
	hipGetDeviceCount(&device_count);
	for (i = 0; i < device_count; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		std::cout << "Running on GPU " << i << " (" << properties.name << ")" << std::endl;
	}

	if (argc < 4){
		printf("Usage :\t%s filter inputfilename.bmp ouputfilename.bmp\n", argv[0]);
		exit(-1);
	}
	// Получаем параметры - имена файлов

	char *filter = argv[1];
	char *inputFileName = argv[2];
	char *outputFileName = argv[3];
	int gridSize = (argc>4)?atoi(argv[4]):0;
	int blockSize = (argc>5)?atoi(argv[5]):0;

	printf("Title :\t%s\n", title);
	printf("Description :\t%s\n", description);
	printf("Filter :\t%s\n", filter);
	printf("Input file name :\t%s\n", inputFileName);
	printf("Output file name :\t%s\n", outputFileName);

	FILE *file = fopen(inputFileName, "rb");
	if (!file) { 
		fprintf(stderr, "Open file error (%s)\n", inputFileName); 
		fflush(stderr); 
		exit(-1); 
	}
	fseek(file, 0L, SEEK_END);
	long size = ftell(file);
	unsigned char *buffer = (unsigned char *)malloc((size_t)size);
	fseek(file, 0L, SEEK_SET);
	fread((void*)buffer, (size_t)1, (size_t)size, file);
	fclose(file);

	uint32_t width = *(uint32_t *)&buffer[BMPWIDTH];
	uint32_t height = *(uint32_t *)&buffer[BMPHEIGHT];
	uint32_t file_size = *(uint32_t *)&buffer[BMPFILESIZE];
	uint32_t offset = *(uint32_t *)&buffer[BMPOFFSET];
	uint16_t bits_per_pixel = *(uint16_t *)&buffer[BMPBITSPERPIXEL];
	uint16_t bytes_per_pixel = ((int)((bits_per_pixel+7)/8));
	uint32_t bytes_per_line = ((int)((bits_per_pixel * width+31)/32))*4; // http://en.wikipedia.org/wiki/BMP_file_format

	printf("BMP image size :\t%ld x %ld\n", width, height);
	printf("BMP file size :\t%ld\n", file_size);
	printf("BMP pixels offset :\t%ld\n", offset);
	printf("BMP bits per pixel :\t%d\n", bits_per_pixel);
	printf("BMP bytes per pixel :\t%d\n", bytes_per_pixel);
	printf("BMP bytes per line :\t%d\n", bytes_per_line);

	uint8_t *pixels =(uint8_t *)&buffer[offset];
	uint8_t *pixel;
	uint32_t x, y;

	/* выделение памяти под байтовые цветовые каналы */
	int count = width*height;
	unsigned int* channel[3];
	for (j = 0; j < 3; j++)
		channel[j] = (unsigned int*)malloc(count*sizeof(unsigned int));

	int pos = 0;
	for (y = 0; y < height; y++) {
		for (x = 0; x < width; x++) {
			pixel = &pixels[y*bytes_per_line+x*bytes_per_pixel];
			channel[0][pos] = (unsigned int)pixel[0];
			channel[1][pos] = (unsigned int)pixel[1];
			channel[2][pos] = (unsigned int)pixel[2];
			pos++;
		}
	}

	clock_t t1, t2;
	t1 = clock();

	host_nonefilter(gridSize, blockSize, channel[0], channel[1], channel[2], width, height);

	t2 = clock();
	double seconds = ((double)(t2-t1))/CLOCKS_PER_SEC;
	printf("Execution time :\t%le\n", seconds);

	pos = 0;
	for (y = 0; y < height; y++) {
		for (x = 0; x < width; x++) {
			pixel = &pixels[y*bytes_per_line+x*bytes_per_pixel];
			pixel[0] = (uint8_t)channel[0][pos];
			pixel[1] = (uint8_t)channel[1][pos];
			pixel[2] = (uint8_t)channel[2][pos];
			pos++;
		}
	}
	/* высвобождаем массивы */
	for (j = 0; j < 3; j++)
		free(channel[j]);

	/* выводим результаты */
	file = fopen(outputFileName, "wb");
	if (!file) {
		fprintf(stderr, "Open file error (%s)\n", outputFileName); 
		fflush(stderr);
		exit(-1);
	}
	fwrite((void*)buffer, (size_t)1, (size_t)size, file);
	printf("Output file size :\t%ld\n", size);
	free(buffer);
	fclose(file);

	hipDeviceReset();

	exit(0);
}