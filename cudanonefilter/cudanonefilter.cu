﻿char *title = "none filtering";
char *description = "none filtering";
/*
Box-фильтр представляет собой так называемую четырехточечную свертку. 
В этом фильтре новое значение пикселя вычисляется, как среднее значение 
его четырех соседних пикселей: верхнего, нижнего, левого и правого.
Вернее рассматривается матрица 3х3 и вычисляется взвешенная сумма соседних элементов.
*/

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <time.h>
#include <hip/hip_runtime.h>


#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define assert( bool ) 

#define BUFFER_SIZE 4096


__global__ void global_nonefilter(
	unsigned int *input,
	unsigned int *output,
	int width, 
	int height)
{
	for (int id = blockDim.x*blockIdx.x + threadIdx.x;
		id < width*height;
		id += blockDim.x*gridDim.x) {
		output[id] = input[id];
	}
}


__host__ void host_nonefilter(int gridSize, int blockSize, unsigned int *r, unsigned int *g, unsigned int *b, int width, int height)
{
	unsigned int *channel[3];
	channel[0] = r;
	channel[1] = g;
	channel[2] = b;

	// channel - массив данных
	// matrix - коэффициенты
	// width - ширина
	// height - высота

	hipError_t err;
	unsigned int *device_input;
	unsigned int *device_output;

	err = hipMalloc((void**)&device_input, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_output, width*height*sizeof(unsigned int));

	int blocks = (gridSize > 0)? gridSize : min(15, (int)sqrt(width*height));
	int threads = (blockSize > 0)? blockSize : min(15, (int)sqrt(width*height));

	for(int j=0; j<3; j++) {
		hipMemcpy(device_input, channel[j], width*height*sizeof(unsigned int), hipMemcpyHostToDevice);

		global_nonefilter <<< blocks, threads >>>(device_input, device_output, width, height);

		hipMemcpy(channel[j], device_output, width*height*sizeof(unsigned int), hipMemcpyDeviceToHost);
	}

	// Освобождаем память на устройстве
	hipFree((void*)device_input);
	hipFree((void*)device_output);
	
	err = err;
}


int main(int argc, char* argv[])
{
	int j, k;
	unsigned char buffer[BUFFER_SIZE];

	std::cout << title << std::endl;

	// Find/set the device.
	int device_count = 0;
	hipGetDeviceCount(&device_count);
	for (int i = 0; i < device_count; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		std::cout << "Running on GPU " << i << " (" << properties.name << ")" << std::endl;
	}

	if (argc < 8){
		printf("Usage :\t%s filter width height inputfilename.bin ouputfilename.bin\n", argv[0]);
		exit(-1);
	}

	unsigned int* channel[3];
	// Получаем параметры - имена файлов

	char *filter = argv[1];
	int width = atoi(argv[2]);
	int height = atoi(argv[3]);
	char *inputFileName = argv[4];
	char *outputFileName = argv[5];
	int gridSize = atoi(argv[6]);
	int blockSize = atoi(argv[7]);

	int count = width*height;

	printf("Title :\t%s\n", title);
	printf("Description :\t%s\n", description);
	printf("Filter :\t%s\n", filter);
	printf("Size :\t%d %d\n", width, height);
	printf("Input file name :\t%s\n", inputFileName);
	printf("Output file name :\t%s\n", outputFileName);

	/* выделение памяти под байтовые цветовые каналы */
	for (j = 0; j < 3; j++)
		channel[j] = (unsigned int*)malloc(width*height*sizeof(unsigned int));

	FILE *file = fopen(inputFileName, "rb");
	if (!file) {
		fprintf(stderr, "Open file error (%s)\n", inputFileName); fflush(stderr);
		exit(-1);
	}

	for (int pos = 0; pos < count; pos += (int)(BUFFER_SIZE / 3)) {
		int size = min((int)(BUFFER_SIZE / 3), (count - pos));
		fread(buffer, (size_t)3, (size_t)size, file);
		for (j = 0; j < 3; j++)
			for (k = 0; k < size; k++)
				channel[j][pos + k] = (unsigned int)buffer[3 * k + j];
	}

	fclose(file);

	host_nonefilter(gridSize, blockSize, channel[0], channel[1], channel[2], width, height);

	/* выводим результаты */
	file = fopen(outputFileName, "wb");
	if (!file) {
		fprintf(stderr, "Open file error (%s)\n", outputFileName); fflush(stderr);
		exit(-1);
	}

	for (int pos = 0; pos < count; pos += (int)(BUFFER_SIZE / 3)) {
		int size = min((int)(BUFFER_SIZE / 3), (count - pos));
		for (j = 0; j < 3; j++)
			for (k = 0; k < size; k++)
				buffer[3 * k + j] = (unsigned char)channel[j][pos + k];
		fwrite(buffer, (size_t)3, (size_t)size, file);
	}

	fclose(file);

	// Высвобождаем массив
	for (j = 0; j < 3; j++)
		free(channel[j]);

	hipDeviceReset();

	exit(0);
}