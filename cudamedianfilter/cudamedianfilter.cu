﻿char *title = "Median Filter";
char *description = "Median Filter";

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <time.h>
#include <hip/hip_runtime.h>


#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define assert( bool ) 

#define BUFFER_SIZE 4096

__global__ void global_medianfilter(
	unsigned int *input,
	unsigned int *output,
	unsigned int *buffer,
	int Nh,
	int width, 
	int height)
{
	for (int id = blockDim.x*blockIdx.x + threadIdx.x;
		id < width*height;
		id += blockDim.x*gridDim.x) {
		int x = id % width;
		int y = id / width;
	
		int k = 0;
		for (int dx = -Nh; dx <= Nh; dx++) {
			for (int dy = -Nh; dy <= Nh; dy++) {
				if (x + dx < 0) continue;
				if (x + dx >= width) continue;
				if (y + dy < 0) continue;
				if (y + dy >= height) continue;
				buffer[(2 * Nh + 1) *(2 * Nh + 1)*(blockDim.x*blockIdx.x + threadIdx.x) + k++] = input[id + dx + width*dy];
			}
		}
		for (int i = (2 * Nh + 1) *(2 * Nh + 1)*(blockDim.x*blockIdx.x + threadIdx.x); i < (2 * Nh + 1) *(2 * Nh + 1)*(blockDim.x*blockIdx.x + threadIdx.x) + k - 1; i++) {
			for (int j = i + 1; j < (2 * Nh + 1) *(2 * Nh + 1)*(blockDim.x*blockIdx.x + threadIdx.x) + k; j++) {
				if (buffer[i] > buffer[j]) {
					unsigned int tmp = buffer[i];
					buffer[i] = buffer[j];
					buffer[j] = tmp;
				}
			}
		}
		output[id] = buffer[(2 * Nh + 1) *(2 * Nh + 1)*(blockDim.x*blockIdx.x + threadIdx.x) + (k >> 1)];
	}
}

__host__ void host_medianfilter(int gridSize, int blockSize, unsigned int *r, unsigned int *g, unsigned int *b, int Nh, int width, int height)
{
	unsigned int *channel[3];
	channel[0] = r;
	channel[1] = g;
	channel[2] = b;

	// channel - массив данных
	// Nh - полуразмер
	// width - ширина
	// height - высота

	hipError_t err;
	unsigned int *device_input;
	unsigned int *device_output;
	unsigned int *device_buffer;

	int blocks = (gridSize > 0)? gridSize : min(15, (int)sqrt(width*height));
	int threads = (blockSize > 0)? blockSize : min(15, (int)sqrt(width*height));

	err = hipMalloc((void**)&device_input, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_output, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_buffer, blocks*threads*(2 * Nh + 1) *(2 * Nh + 1) * sizeof(int));

	for(int j=0; j<3; j++) {
		hipMemcpy(device_input, channel[j], width*height*sizeof(unsigned int), hipMemcpyHostToDevice);

		global_medianfilter <<< blocks, threads >>>(device_input, device_output, device_buffer, Nh, width, height);

		hipMemcpy(channel[j], device_output, width*height*sizeof(unsigned int), hipMemcpyDeviceToHost);
	}

	// Освобождаем память на устройстве
	hipFree((void*)device_input);
	hipFree((void*)device_output);
	hipFree((void*)device_buffer);
	
	err = err;
}



int main(int argc, char* argv[])
{
	int j, k;
	unsigned char buffer[BUFFER_SIZE];

	std::cout << title << std::endl;

	// Find/set the device.
	int device_count = 0;
	hipGetDeviceCount(&device_count);
	for (int i = 0; i < device_count; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		std::cout << "Running on GPU " << i << " (" << properties.name << ")" << std::endl;
	}

	if (argc < 9){
		printf("Usage :\t%s filter Nh width height inputfilename.bin ouputfilename.bin\n", argv[0]);
		exit(-1);
	}

	unsigned int* channel[3];
	// Получаем параметры - имена файлов

	char *filter = argv[1];
	int step = atoi(argv[2]);
	int width = atoi(argv[3]);
	int height = atoi(argv[4]);
	char *inputFileName = argv[5];
	char *outputFileName = argv[6];
	int gridSize = atoi(argv[7]);
	int blockSize = atoi(argv[8]);

	printf("Title :\t%s\n", title);
	printf("Description :\t%s\n", description);
	printf("Filter :\t%s\n", filter);
	printf("Step :\t%d\n", step);
	printf("Size :\t%d %d\n", width, height);
	printf("Input file name :\t%s\n", inputFileName);
	printf("Output file name :\t%s\n", outputFileName);

	int count = width*height;

	/* Расчёт размеров полей в зависимости от переданного параметра */
	int N = step % 2 == 0 ? step += 1 : step;
	int Nh = N >> 1;

	/* выделение памяти под байтовые цветовые каналы */
	for (j = 0; j < 3; j++)
		channel[j] = (unsigned int*)malloc(width*height*sizeof(unsigned int));

	FILE *file = fopen(inputFileName, "rb");
	if (!file) {
		fprintf(stderr, "Open file error (%s)\n", inputFileName); fflush(stderr);
		exit(-1);
	}

	for (int pos = 0; pos < count; pos += (int)(BUFFER_SIZE / 3)) {
		int size = min((int)(BUFFER_SIZE / 3), (count - pos));
		fread(buffer, (size_t)3, (size_t)size, file);
		for (j = 0; j < 3; j++)
			for (k = 0; k < size; k++)
				channel[j][pos + k] = (unsigned int)buffer[3 * k + j];
	}

	fclose(file);

	host_medianfilter(gridSize, blockSize, channel[0], channel[1], channel[2], Nh, width, height);

	/* выводим результаты */
	file = fopen(outputFileName, "wb");
	if (!file) {
		fprintf(stderr, "Open file error (%s)\n", outputFileName); fflush(stderr);
		exit(-1);
	}

	for (int pos = 0; pos < count; pos += (int)(BUFFER_SIZE / 3)) {
		int size = min((int)(BUFFER_SIZE / 3), (count - pos));
		for (j = 0; j < 3; j++)
			for (k = 0; k < size; k++)
				buffer[3 * k + j] = (unsigned char)channel[j][pos + k];
		fwrite(buffer, (size_t)3, (size_t)size, file);
	}

	fclose(file);

	// Высвобождаем массив
	for (j = 0; j < 3; j++)
		free(channel[j]);

	hipDeviceReset();

	exit(0);
}