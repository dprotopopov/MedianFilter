﻿char *title = "Median Filter";
char *description = "Median Filter";

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>


#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#ifndef uint8_t
typedef unsigned char uint8_t;
#endif

#ifndef uint16_t
typedef unsigned short uint16_t;
#endif

#ifndef uint32_t
typedef unsigned int uint32_t;
#endif

// http://en.wikipedia.org/wiki/BMP_file_format
#define BMPMAGIC	0x00
#define BMPFILESIZE	0x02
#define BMPOFFSET	0x0A
#define BMPDIBSISE	0x0E
#define BMPWIDTH	0x12
#define BMPHEIGHT	0x16
#define BMPBITSPERPIXEL	0x1C

typedef struct {
	uint8_t magic[2];   /* the magic number used to identify the BMP file:
						0x42 0x4D (Hex code points for B and M).
						The following entries are possible:
						BM - Windows 3.1x, 95, NT, ... etc
						BA - OS/2 Bitmap Array
						CI - OS/2 Color Icon
						CP - OS/2 Color Pointer
						IC - OS/2 Icon
						PT - OS/2 Pointer. */
	uint32_t filesz;    /* the size of the BMP file in bytes */
	uint16_t creator1;  /* reserved. */
	uint16_t creator2;  /* reserved. */
	uint32_t offset;    /* the offset, i.e. starting address,
						of the byte where the bitmap data can be found. */
} bmp_header_t;

typedef struct {
	uint32_t header_sz;     /* the size of this header (40 bytes) */
	uint32_t width;         /* the bitmap width in pixels */
	uint32_t height;        /* the bitmap height in pixels */
	uint16_t nplanes;       /* the number of color planes being used.
							Must be set to 1. */
	uint16_t depth;         /* the number of bits per pixel,
							which is the color depth of the image.
							Typical values are 1, 4, 8, 16, 24 and 32. */
	uint32_t compress_type; /* the compression method being used.
							See also bmp_compression_method_t. */
	uint32_t bmp_bytesz;    /* the image size. This is the size of the raw bitmap
							data (see below), and should not be confused
							with the file size. */
	uint32_t hres;          /* the horizontal resolution of the image.
							(pixel per meter) */
	uint32_t vres;          /* the vertical resolution of the image.
							(pixel per meter) */
	uint32_t ncolors;       /* the number of colors in the color palette,
							or 0 to default to 2<sup><i>n</i></sup>. */
	uint32_t nimpcolors;    /* the number of important colors used,
							or 0 when every color is important;
							generally ignored. */
} bmp_dib_v3_header_t;


__global__ void global_medianfilter(
	unsigned int *input,
	unsigned int *output,
	unsigned int *buffer,
	int Nh,
	int width, 
	int height)
{
	int bufferId = (2 * Nh + 1) *(2 * Nh + 1)*(blockDim.x*blockIdx.x + threadIdx.x);
	for (int id = blockDim.x*blockIdx.x + threadIdx.x;
		id < width*height;
		id += blockDim.x*gridDim.x) {
			int x = id % width;
			int y = id / width;

			int k = 0;
			for (int dx = -Nh; dx <= Nh; dx++) {
				for (int dy = -Nh; dy <= Nh; dy++) {
					if (x + dx < 0) continue;
					if (x + dx >= width) continue;
					if (y + dy < 0) continue;
					if (y + dy >= height) continue;
					buffer[bufferId + k++] = input[id + dx + width*dy];
				}
			}
			for (int i = bufferId; i <bufferId + k - 1; i++) {
				for (int j = i + 1; j < bufferId + k; j++) {
					if (buffer[i] > buffer[j]) {
						unsigned int tmp = buffer[i];
						buffer[i] = buffer[j];
						buffer[j] = tmp;
					}
				}
			}
			output[id] = buffer[bufferId + (k >> 1)];
	}
}

__host__ void host_medianfilter(int gridSize, int blockSize, unsigned int *r, unsigned int *g, unsigned int *b, int Nh, int width, int height)
{
	unsigned int *channel[3] = { r,g,b };

	// channel - массив данных
	// Nh - полуразмер
	// width - ширина
	// height - высота

	hipError_t err;
	unsigned int *device_input;
	unsigned int *device_output;
	unsigned int *device_buffer;

	int blocks = (gridSize > 0)? gridSize : min(15, (int)sqrt(width*height));
	int threads = (blockSize > 0)? blockSize : min(15, (int)sqrt(width*height));

	err = hipMalloc((void**)&device_input, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_output, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_buffer, blocks*threads*(2 * Nh + 1) *(2 * Nh + 1) * sizeof(int));

	for(int j=0; j<3; j++) {
		hipMemcpy(device_input, channel[j], width*height*sizeof(unsigned int), hipMemcpyHostToDevice);

		global_medianfilter <<< blocks, threads >>>(device_input, device_output, device_buffer, Nh, width, height);

		hipMemcpy(channel[j], device_output, width*height*sizeof(unsigned int), hipMemcpyDeviceToHost);
	}

	// Освобождаем память на устройстве
	hipFree((void*)device_input);
	hipFree((void*)device_output);
	hipFree((void*)device_buffer);

	err = err;
}

int main(int argc, char* argv[])
{
	int i, j;

	std::cout << title << std::endl;

	// Find/set the device.
	int device_count = 0;
	hipGetDeviceCount(&device_count);
	for (i = 0; i < device_count; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		std::cout << "Running on GPU " << i << " (" << properties.name << ")" << std::endl;
	}

	if (argc < 5){
		printf("Usage :\t%s filter Nh inputfilename.bmp ouputfilename.bmp\n", argv[0]);
		exit(-1);
	}

	// Получаем параметры - имена файлов

	char *filter = argv[1];
	int step = atoi(argv[2]);
	char *inputFileName = argv[3];
	char *outputFileName = argv[4];
	int gridSize = (argc>5)?atoi(argv[5]):0;
	int blockSize = (argc>6)?atoi(argv[6]):0;

	printf("Title :\t%s\n", title);
	printf("Description :\t%s\n", description);
	printf("Filter :\t%s\n", filter);
	printf("Step :\t%d\n", step);
	printf("Input file name :\t%s\n", inputFileName);
	printf("Output file name :\t%s\n", outputFileName);

	/* Расчёт размеров полей в зависимости от переданного параметра */
	int N = step % 2 == 0 ? step += 1 : step;
	int Nh = N >> 1;

	FILE *file = fopen(inputFileName, "rb");
	if (!file) { 
		fprintf(stderr, "Open file error (%s)\n", inputFileName); 
		fflush(stderr); 
		exit(-1); 
	}
	fseek(file, 0L, SEEK_END);
	long size = ftell(file);
	unsigned char *buffer = (unsigned char *)malloc((size_t)size);
	fseek(file, 0L, SEEK_SET);
	fread((void*)buffer, (size_t)1, (size_t)size, file);
	fclose(file);

	uint32_t width = *(uint32_t *)&buffer[BMPWIDTH];
	uint32_t height = *(uint32_t *)&buffer[BMPHEIGHT];
	uint32_t file_size = *(uint32_t *)&buffer[BMPFILESIZE];
	uint32_t offset = *(uint32_t *)&buffer[BMPOFFSET];
	uint16_t bits_per_pixel = *(uint16_t *)&buffer[BMPBITSPERPIXEL];
	uint16_t bytes_per_pixel = ((int)((bits_per_pixel+7)/8));
	uint32_t bytes_per_line = ((int)((bits_per_pixel * width+31)/32))*4; // http://en.wikipedia.org/wiki/BMP_file_format

	printf("BMP image size :\t%ld x %ld\n", width, height);
	printf("BMP file size :\t%ld\n", file_size);
	printf("BMP pixels offset :\t%ld\n", offset);
	printf("BMP bits per pixel :\t%d\n", bits_per_pixel);
	printf("BMP bytes per pixel :\t%d\n", bytes_per_pixel);
	printf("BMP bytes per line :\t%d\n", bytes_per_line);

	uint8_t *pixels =(uint8_t *)&buffer[offset];
	uint8_t *pixel;
	uint32_t x, y;

	/* выделение памяти под байтовые цветовые каналы */
	int count = width*height;
	unsigned int* channel[3];
	for (j = 0; j < 3; j++)
		channel[j] = (unsigned int*)malloc(count*sizeof(unsigned int));

	int pos = 0;
	for (y = 0; y < height; y++) {
		for (x = 0; x < width; x++) {
			pixel = &pixels[y*bytes_per_line+x*bytes_per_pixel];
			channel[0][pos] = (unsigned int)pixel[0];
			channel[1][pos] = (unsigned int)pixel[1];
			channel[2][pos] = (unsigned int)pixel[2];
			pos++;
		}
	}

	clock_t t1, t2;
	t1 = clock();

	host_medianfilter(gridSize, blockSize, channel[0], channel[1], channel[2], Nh, width, height);

	t2 = clock();
	double seconds = ((double)(t2-t1))/CLOCKS_PER_SEC;
	printf("Execution time :\t%le\n", seconds);

	pos = 0;
	for (y = 0; y < height; y++) {
		for (x = 0; x < width; x++) {
			pixel = &pixels[y*bytes_per_line+x*bytes_per_pixel];
			pixel[0] = (uint8_t)channel[0][pos];
			pixel[1] = (uint8_t)channel[1][pos];
			pixel[2] = (uint8_t)channel[2][pos];
			pos++;
		}
	}
	/* высвобождаем массивы */
	for (j = 0; j < 3; j++)
		free(channel[j]);

	/* выводим результаты */
	file = fopen(outputFileName, "wb");
	if (!file) {
		fprintf(stderr, "Open file error (%s)\n", outputFileName); 
		fflush(stderr);
		exit(-1);
	}
	fwrite((void*)buffer, (size_t)1, (size_t)size, file);
	printf("Output file size :\t%ld\n", size);
	free(buffer);
	fclose(file);

	hipDeviceReset();

	exit(0);
}