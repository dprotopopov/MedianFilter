﻿char *title = "box filtering";
char *description = "box filtering";
/*
Box-фильтр представляет собой так называемую четырехточечную свертку. 
В этом фильтре новое значение пикселя вычисляется, как среднее значение 
его четырех соседних пикселей: верхнего, нижнего, левого и правого.
Вернее рассматривается матрица 3х3 и вычисляется взвешенная сумма соседних элементов.
*/

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>


#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#ifndef uint8_t
typedef unsigned char uint8_t;
#endif

#ifndef uint16_t
typedef unsigned short uint16_t;
#endif

#ifndef uint32_t
typedef unsigned int uint32_t;
#endif

// http://en.wikipedia.org/wiki/BMP_file_format
#define BMPMAGIC	0x00
#define BMPFILESIZE	0x02
#define BMPOFFSET	0x0A
#define BMPDIBSISE	0x0E
#define BMPWIDTH	0x12
#define BMPHEIGHT	0x16
#define BMPBITSPERPIXEL	0x1C

typedef struct {
	uint8_t magic[2];   /* the magic number used to identify the BMP file:
						0x42 0x4D (Hex code points for B and M).
						The following entries are possible:
						BM - Windows 3.1x, 95, NT, ... etc
						BA - OS/2 Bitmap Array
						CI - OS/2 Color Icon
						CP - OS/2 Color Pointer
						IC - OS/2 Icon
						PT - OS/2 Pointer. */
	uint32_t filesz;    /* the size of the BMP file in bytes */
	uint16_t creator1;  /* reserved. */
	uint16_t creator2;  /* reserved. */
	uint32_t offset;    /* the offset, i.e. starting address,
						of the byte where the bitmap data can be found. */
} bmp_header_t;

typedef struct {
	uint32_t header_sz;     /* the size of this header (40 bytes) */
	uint32_t width;         /* the bitmap width in pixels */
	uint32_t height;        /* the bitmap height in pixels */
	uint16_t nplanes;       /* the number of color planes being used.
							Must be set to 1. */
	uint16_t depth;         /* the number of bits per pixel,
							which is the color depth of the image.
							Typical values are 1, 4, 8, 16, 24 and 32. */
	uint32_t compress_type; /* the compression method being used.
							See also bmp_compression_method_t. */
	uint32_t bmp_bytesz;    /* the image size. This is the size of the raw bitmap
							data (see below), and should not be confused
							with the file size. */
	uint32_t hres;          /* the horizontal resolution of the image.
							(pixel per meter) */
	uint32_t vres;          /* the vertical resolution of the image.
							(pixel per meter) */
	uint32_t ncolors;       /* the number of colors in the color palette,
							or 0 to default to 2<sup><i>n</i></sup>. */
	uint32_t nimpcolors;    /* the number of important colors used,
							or 0 when every color is important;
							generally ignored. */
} bmp_dib_v3_header_t;

__global__ void global_boxfilter(
	unsigned int *input,
	unsigned int *output,
	int *matrix,
	int width, 
	int height)
{
	for (int id = blockDim.x*blockIdx.x + threadIdx.x;
		id < width*height;
		id += blockDim.x*gridDim.x) {
			int x = id % width;
			int y = id / width;

			int s1 = 0;
			int s2 = 0;
			for (int dx = -1; dx <= 1; dx++) {
				for (int dy = -1; dy <= 1; dy++) {
					if (x + dx < 0) continue;
					if (x + dx >= width) continue;
					if (y + dy < 0) continue;
					if (y + dy >= height) continue;
					int value = (int)(unsigned int)input[id + dx + width*dy];
					s1 += value*matrix[(1 + dx) + 3 * (1 + dy)];
					s2 += matrix[(1 + dx) + 3 * (1 + dy)];
				}
			}
			output[id] = (unsigned int)(s1 / s2);
	}
}


__host__ void host_boxfilter(int gridSize, int blockSize, unsigned int *r, unsigned int *g, unsigned int *b, int *matrix, int width, int height)
{
	unsigned int *channel[3] = { r,g,b };

	// channel - массив данных
	// matrix - коэффициенты
	// width - ширина
	// height - высота

	hipError_t err;
	unsigned int *device_input;
	unsigned int *device_output;
	int *device_matrix;

	err = hipMalloc((void**)&device_input, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_output, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_matrix, 9 * sizeof(int));
	hipMemcpy(device_matrix, matrix, 9*sizeof(int), hipMemcpyHostToDevice);

	int blocks = (gridSize > 0)? gridSize : min(15, (int)sqrt(width*height));
	int threads = (blockSize > 0)? blockSize : min(15, (int)sqrt(width*height));

	for(int j=0; j<3; j++) {
		hipMemcpy(device_input, channel[j], width*height*sizeof(unsigned int), hipMemcpyHostToDevice);

		global_boxfilter <<< blocks, threads >>>(device_input, device_output, device_matrix, width, height);

		hipMemcpy(channel[j], device_output, width*height*sizeof(unsigned int), hipMemcpyDeviceToHost);
	}

	// Освобождаем память на устройстве
	hipFree((void*)device_input);
	hipFree((void*)device_output);
	hipFree((void*)device_matrix);

	err = err;
}


int main(int argc, char* argv[])
{
	int i, j;

	std::cout << title << std::endl;

	// Find/set the device.
	int device_count = 0;
	hipGetDeviceCount(&device_count);
	for (i = 0; i < device_count; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		std::cout << "Running on GPU " << i << " (" << properties.name << ")" << std::endl;
	}

	if (argc < 13){
		printf("Usage :\t%s filter matrix[0][0] ... matrix[2][2] inputfilename.bmp ouputfilename.bmp\n", argv[0]);
		exit(-1);
	}

	int matrix[9];

	// Получаем параметры - имена файлов

	char *filter = argv[1];
	matrix[0] = atoi(argv[2 + 3 * 0 + 0]);
	matrix[1] = atoi(argv[2 + 3 * 0 + 1]);
	matrix[2] = atoi(argv[2 + 3 * 0 + 2]);
	matrix[3] = atoi(argv[2 + 3 * 1 + 0]);
	matrix[4] = atoi(argv[2 + 3 * 1 + 1]);
	matrix[5] = atoi(argv[2 + 3 * 1 + 2]);
	matrix[6] = atoi(argv[2 + 3 * 2 + 0]);
	matrix[7] = atoi(argv[2 + 3 * 2 + 1]);
	matrix[8] = atoi(argv[2 + 3 * 2 + 2]);
	char *inputFileName = argv[11];
	char *outputFileName = argv[12];
	int gridSize = (argc>13)?atoi(argv[13]):0;
	int blockSize = (argc>14)?atoi(argv[14]):0;

	printf("Title :\t%s\n", title);
	printf("Description :\t%s\n", description);
	printf("Filter :\t%s\n", filter);
	printf("Input file name :\t%s\n", inputFileName);
	printf("Output file name :\t%s\n", outputFileName);

	FILE *file = fopen(inputFileName, "rb");
	if (!file) { 
		fprintf(stderr, "Open file error (%s)\n", inputFileName); 
		fflush(stderr); 
		exit(-1); 
	}
	fseek(file, 0L, SEEK_END);
	long size = ftell(file);
	unsigned char *buffer = (unsigned char *)malloc((size_t)size);
	fseek(file, 0L, SEEK_SET);
	fread((void*)buffer, (size_t)1, (size_t)size, file);
	fclose(file);

	uint32_t width = *(uint32_t *)&buffer[BMPWIDTH];
	uint32_t height = *(uint32_t *)&buffer[BMPHEIGHT];
	uint32_t file_size = *(uint32_t *)&buffer[BMPFILESIZE];
	uint32_t offset = *(uint32_t *)&buffer[BMPOFFSET];
	uint16_t bits_per_pixel = *(uint16_t *)&buffer[BMPBITSPERPIXEL];
	uint16_t bytes_per_pixel = ((int)((bits_per_pixel+7)/8));
	uint32_t bytes_per_line = ((int)((bits_per_pixel * width+31)/32))*4; // http://en.wikipedia.org/wiki/BMP_file_format

	printf("BMP image size :\t%ld x %ld\n", width, height);
	printf("BMP file size :\t%ld\n", file_size);
	printf("BMP pixels offset :\t%ld\n", offset);
	printf("BMP bits per pixel :\t%d\n", bits_per_pixel);
	printf("BMP bytes per pixel :\t%d\n", bytes_per_pixel);
	printf("BMP bytes per line :\t%d\n", bytes_per_line);

	uint8_t *pixels =(uint8_t *)&buffer[offset];
	uint8_t *pixel;
	uint32_t x, y;

	/* выделение памяти под байтовые цветовые каналы */
	int count = width*height;
	unsigned int* channel[3];
	for (j = 0; j < 3; j++)
		channel[j] = (unsigned int*)malloc(count*sizeof(unsigned int));

	int pos = 0;
	for (y = 0; y < height; y++) {
		for (x = 0; x < width; x++) {
			pixel = &pixels[y*bytes_per_line+x*bytes_per_pixel];
			channel[0][pos] = (unsigned int)pixel[0];
			channel[1][pos] = (unsigned int)pixel[1];
			channel[2][pos] = (unsigned int)pixel[2];
			pos++;
		}
	}

	clock_t t1, t2;
	t1 = clock();

	host_boxfilter(gridSize, blockSize, channel[0], channel[1], channel[2], matrix, width, height);

	t2 = clock();
	double seconds = ((double)(t2-t1))/CLOCKS_PER_SEC;
	printf("Execution time :\t%le\n", seconds);

	pos = 0;
	for (y = 0; y < height; y++) {
		for (x = 0; x < width; x++) {
			pixel = &pixels[y*bytes_per_line+x*bytes_per_pixel];
			pixel[0] = (uint8_t)channel[0][pos];
			pixel[1] = (uint8_t)channel[1][pos];
			pixel[2] = (uint8_t)channel[2][pos];
			pos++;
		}
	}
	/* высвобождаем массивы */
	for (j = 0; j < 3; j++)
		free(channel[j]);

	/* выводим результаты */
	file = fopen(outputFileName, "wb");
	if (!file) {
		fprintf(stderr, "Open file error (%s)\n", outputFileName); 
		fflush(stderr);
		exit(-1);
	}
	fwrite((void*)buffer, (size_t)1, (size_t)size, file);
	printf("Output file size :\t%ld\n", size);
	free(buffer);
	fclose(file);

	hipDeviceReset();

	exit(0);
}