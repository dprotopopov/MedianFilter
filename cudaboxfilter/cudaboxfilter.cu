﻿char *title = "box filtering";
char *description = "box filtering";
/*
Box-фильтр представляет собой так называемую четырехточечную свертку. 
В этом фильтре новое значение пикселя вычисляется, как среднее значение 
его четырех соседних пикселей: верхнего, нижнего, левого и правого.
Вернее рассматривается матрица 3х3 и вычисляется взвешенная сумма соседних элементов.
*/

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <time.h>
#include <hip/hip_runtime.h>


#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define assert( bool ) 

#define BUFFER_SIZE 4096


__global__ void global_boxfilter(
	unsigned int *input,
	unsigned int *output,
	int *matrix,
	int width, 
	int height)
{
	for (int id = blockDim.x*blockIdx.x + threadIdx.x;
		id < width*height;
		id += blockDim.x*gridDim.x) {
		int x = id % width;
		int y = id / width;
	
		int s1 = 0;
		int s2 = 0;
		for (int dx = -1; dx <= 1; dx++) {
			for (int dy = -1; dy <= 1; dy++) {
				if (x + dx < 0) continue;
				if (x + dx >= width) continue;
				if (y + dy < 0) continue;
				if (y + dy >= height) continue;
				int value = (int)(unsigned int)input[id + dx + width*dy];
				s1 += value*matrix[(1 + dx) + 3 * (1 + dy)];
				s2 += matrix[(1 + dx) + 3 * (1 + dy)];
			}
		}
		output[id] = (unsigned int)(s1 / s2);
	}
}


__host__ void host_boxfilter(int gridSize, int blockSize, unsigned int *r, unsigned int *g, unsigned int *b, int *matrix, int width, int height)
{
	unsigned int *channel[3];
	channel[0] = r;
	channel[1] = g;
	channel[2] = b;

	// channel - массив данных
	// matrix - коэффициенты
	// width - ширина
	// height - высота

	hipError_t err;
	unsigned int *device_input;
	unsigned int *device_output;
	int *device_matrix;

	err = hipMalloc((void**)&device_input, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_output, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_matrix, 9 * sizeof(int));
	hipMemcpy(device_matrix, matrix, 9*sizeof(int), hipMemcpyHostToDevice);

	int blocks = (gridSize > 0)? gridSize : min(15, (int)sqrt(width*height));
	int threads = (blockSize > 0)? blockSize : min(15, (int)sqrt(width*height));

	for(int j=0; j<3; j++) {
		hipMemcpy(device_input, channel[j], width*height*sizeof(unsigned int), hipMemcpyHostToDevice);

		global_boxfilter <<< blocks, threads >>>(device_input, device_output, device_matrix, width, height);

		hipMemcpy(channel[j], device_output, width*height*sizeof(unsigned int), hipMemcpyDeviceToHost);
	}

	// Освобождаем память на устройстве
	hipFree((void*)device_input);
	hipFree((void*)device_output);
	hipFree((void*)device_matrix);
	
	err = err;
}


int main(int argc, char* argv[])
{
	int j, k;
	unsigned char buffer[BUFFER_SIZE];

	std::cout << title << std::endl;

	// Find/set the device.
	int device_count = 0;
	hipGetDeviceCount(&device_count);
	for (int i = 0; i < device_count; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		std::cout << "Running on GPU " << i << " (" << properties.name << ")" << std::endl;
	}

	if (argc < 17){
		printf("Usage :\t%s filter matrix[0][0] ... matrix[2][2] width height inputfilename.bin ouputfilename.bin\n", argv[0]);
		exit(-1);
	}

	int matrix[9];
	unsigned int* channel[3];
	// Получаем параметры - имена файлов

	char *filter = argv[1];
	matrix[0] = atoi(argv[2 + 3 * 0 + 0]);
	matrix[1] = atoi(argv[2 + 3 * 0 + 1]);
	matrix[2] = atoi(argv[2 + 3 * 0 + 2]);
	matrix[3] = atoi(argv[2 + 3 * 1 + 0]);
	matrix[4] = atoi(argv[2 + 3 * 1 + 1]);
	matrix[5] = atoi(argv[2 + 3 * 1 + 2]);
	matrix[6] = atoi(argv[2 + 3 * 2 + 0]);
	matrix[7] = atoi(argv[2 + 3 * 2 + 1]);
	matrix[8] = atoi(argv[2 + 3 * 2 + 2]);
	int width = atoi(argv[11]);
	int height = atoi(argv[12]);
	char *inputFileName = argv[13];
	char *outputFileName = argv[14];
	int gridSize = atoi(argv[15]);
	int blockSize = atoi(argv[16]);

	int count = width*height;

	printf("Title :\t%s\n", title);
	printf("Description :\t%s\n", description);
	printf("Filter :\t%s\n", filter);
	printf("Size :\t%d %d\n", width, height);
	printf("Input file name :\t%s\n", inputFileName);
	printf("Output file name :\t%s\n", outputFileName);

	/* выделение памяти под байтовые цветовые каналы */
	for (j = 0; j < 3; j++)
		channel[j] = (unsigned int*)malloc(width*height*sizeof(unsigned int));

	FILE *file = fopen(inputFileName, "rb");
	if (!file) {
		fprintf(stderr, "Open file error (%s)\n", inputFileName); fflush(stderr);
		exit(-1);
	}

	for (int pos = 0; pos < count; pos += (int)(BUFFER_SIZE / 3)) {
		int size = min((int)(BUFFER_SIZE / 3), (count - pos));
		fread(buffer, (size_t)3, (size_t)size, file);
		for (j = 0; j < 3; j++)
			for (k = 0; k < size; k++)
				channel[j][pos + k] = (unsigned int)buffer[3 * k + j];
	}

	fclose(file);

	host_boxfilter(gridSize, blockSize, channel[0], channel[1], channel[2], matrix, width, height);

	/* выводим результаты */
	file = fopen(outputFileName, "wb");
	if (!file) {
		fprintf(stderr, "Open file error (%s)\n", outputFileName); fflush(stderr);
		exit(-1);
	}

	for (int pos = 0; pos < count; pos += (int)(BUFFER_SIZE / 3)) {
		int size = min((int)(BUFFER_SIZE / 3), (count - pos));
		for (j = 0; j < 3; j++)
			for (k = 0; k < size; k++)
				buffer[3 * k + j] = (unsigned char)channel[j][pos + k];
		fwrite(buffer, (size_t)3, (size_t)size, file);
	}

	fclose(file);

	// Высвобождаем массив
	for (j = 0; j < 3; j++)
		free(channel[j]);

	hipDeviceReset();

	exit(0);
}