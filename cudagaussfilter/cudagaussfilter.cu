﻿char *title = "gauss filtering";
char *description = "gauss filtering";

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <time.h>
#include <hip/hip_runtime.h>

#include <math.h>       /* exp */

#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define assert( bool ) 

#define BUFFER_SIZE 4096


__global__ void global_gaussfilter_by_row(
	unsigned int *input,
	unsigned int *output,
	double *matrix,
	int Nh,
	int width,
	int height)
{
	for (int id = blockDim.x*blockIdx.x + threadIdx.x;
		id < width*height;
		id += blockDim.x*gridDim.x) {
		int x = id % width;

		double s1 = 0;
		double s2 = 0;
		for (int dx = -Nh; dx <= Nh; dx++) {
			if (x + dx < 0) continue;
			if (x + dx >= width) continue;
			int value = (double)(unsigned int)input[id + dx];
			s1 += value*matrix[Nh + dx];
			s2 += matrix[Nh + dx];
		}
		output[id] = (unsigned int)(s1 / s2);
	}
}
__global__ void global_gaussfilter_by_col(
	unsigned int *input,
	unsigned int *output,
	double *matrix,
	int Nh,
	int width,
	int height)
{
	for (int id = blockDim.x*blockIdx.x + threadIdx.x;
		id < width*height;
		id += blockDim.x*gridDim.x) {
		int y = id / width;

		double s1 = 0;
		double s2 = 0;
		for (int dy = -Nh; dy <= Nh; dy++) {
			if (y + dy < 0) continue;
			if (y + dy >= height) continue;
			int value = (double)(unsigned int)input[id + width*dy];
			s1 += value*matrix[Nh + dy];
			s2 += matrix[Nh + dy];
		}
		output[id] = (unsigned int)(s1 / s2);
	}
}


__host__ void host_gaussfilter(int gridSize, int blockSize, unsigned int *r, unsigned int *g, unsigned int *b, double *matrix, int Nh, int width, int height)
{
	unsigned int *channel[3];
	channel[0] = r;
	channel[1] = g;
	channel[2] = b;

	// channel - массив данных
	// sigma - коэффициенты
	// width - ширина
	// height - высота

	hipError_t err;
	unsigned int *device_a;
	unsigned int *device_b;
	double *device_matrix;

	err = hipMalloc((void**)&device_a, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_b, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_matrix, (2 * Nh + 1) * sizeof(double));
	hipMemcpy(device_matrix, matrix, (2 * Nh + 1)*sizeof(double), hipMemcpyHostToDevice);

	int blocks = (gridSize > 0)? gridSize : min(15, (int)sqrt(width*height));
	int threads = (blockSize > 0)? blockSize : min(15, (int)sqrt(width*height));

	for(int j=0; j<3; j++) {
		hipMemcpy(device_a, channel[j], width*height*sizeof(unsigned int), hipMemcpyHostToDevice);

		global_gaussfilter_by_row <<< blocks, threads >>>(device_a, device_b, device_matrix, Nh, width, height);
		global_gaussfilter_by_col <<< blocks, threads >>>(device_b, device_a, device_matrix, Nh, width, height);

		hipMemcpy(channel[j], device_a, width*height*sizeof(unsigned int), hipMemcpyDeviceToHost);
	}

	// Освобождаем память на устройстве
	hipFree((void*)device_a);
	hipFree((void*)device_b);
	hipFree((void*)device_matrix);
	
	err = err;
}


int main(int argc, char* argv[])
{
	int j, k;
	unsigned char buffer[BUFFER_SIZE];

	std::cout << title << std::endl;

	// Find/set the device.
	int device_count = 0;
	hipGetDeviceCount(&device_count);
	for (int i = 0; i < device_count; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		std::cout << "Running on GPU " << i << " (" << properties.name << ")" << std::endl;
	}

	if (argc < 9){
		printf("Usage :\t%s filter sigma width height inputfilename.bin ouputfilename.bin\n", argv[0]);
		exit(-1);
	}

	unsigned int* channel[3];

	// Получаем параметры - имена файлов

	char *filter = argv[1];
	double sigma = atof(argv[2]);
	int width = atoi(argv[3]);
	int height = atoi(argv[4]);
	char *inputFileName = argv[5];
	char *outputFileName = argv[6];
	int gridSize = atoi(argv[7]);
	int blockSize = atoi(argv[8]);

	// Nh - полуразмер
	int Nh = (int)(3 * sigma);
	
	double * matrix = (double*)malloc((2 * Nh + 1)*sizeof(double));
	for (int i = -Nh; i <= Nh; i++) matrix[Nh + i] = exp(-(double)i*i / (2.0*sigma*sigma));

	int count = width*height;

	printf("Title :\t%s\n", title);
	printf("Description :\t%s\n", description);
	printf("Filter :\t%s\n", filter);
	printf("Size :\t%d %d\n", width, height);
	printf("Input file name :\t%s\n", inputFileName);
	printf("Output file name :\t%s\n", outputFileName);

	/* выделение памяти под байтовые цветовые каналы */
	for (j = 0; j < 3; j++)
		channel[j] = (unsigned int*)malloc(width*height*sizeof(unsigned int));

	FILE *file = fopen(inputFileName, "rb");
	if (!file) {
		fprintf(stderr, "Open file error (%s)\n", inputFileName); fflush(stderr);
		exit(-1);
	}

	for (int pos = 0; pos < count; pos += (int)(BUFFER_SIZE / 3)) {
		int size = min((int)(BUFFER_SIZE / 3), (count - pos));
		fread(buffer, (size_t)3, (size_t)size, file);
		for (j = 0; j < 3; j++)
			for (k = 0; k < size; k++)
				channel[j][pos + k] = (unsigned int)buffer[3 * k + j];
	}

	fclose(file);

	host_gaussfilter(gridSize, blockSize, channel[0], channel[1], channel[2], matrix, Nh, width, height);

	/* выводим результаты */
	file = fopen(outputFileName, "wb");
	if (!file) {
		fprintf(stderr, "Open file error (%s)\n", outputFileName); fflush(stderr);
		exit(-1);
	}

	for (int pos = 0; pos < count; pos += (int)(BUFFER_SIZE / 3)) {
		int size = min((int)(BUFFER_SIZE / 3), (count - pos));
		for (j = 0; j < 3; j++)
			for (k = 0; k < size; k++)
				buffer[3 * k + j] = (unsigned char)channel[j][pos + k];
		fwrite(buffer, (size_t)3, (size_t)size, file);
	}

	fclose(file);

	// Высвобождаем массив
	for (j = 0; j < 3; j++)
		free(channel[j]);

	hipDeviceReset();

	exit(0);
}