﻿char *title = "gauss filtering";
char *description = "gauss filtering";

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <math.h>       /* exp */

#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#ifndef uint8_t
typedef unsigned char uint8_t;
#endif

#ifndef uint16_t
typedef unsigned short uint16_t;
#endif

#ifndef uint32_t
typedef unsigned int uint32_t;
#endif

// http://en.wikipedia.org/wiki/BMP_file_format
#define BMPMAGIC	0x00
#define BMPFILESIZE	0x02
#define BMPOFFSET	0x0A
#define BMPDIBSISE	0x0E
#define BMPWIDTH	0x12
#define BMPHEIGHT	0x16
#define BMPBITSPERPIXEL	0x1C

typedef struct {
	uint8_t magic[2];   /* the magic number used to identify the BMP file:
						0x42 0x4D (Hex code points for B and M).
						The following entries are possible:
						BM - Windows 3.1x, 95, NT, ... etc
						BA - OS/2 Bitmap Array
						CI - OS/2 Color Icon
						CP - OS/2 Color Pointer
						IC - OS/2 Icon
						PT - OS/2 Pointer. */
	uint32_t filesz;    /* the size of the BMP file in bytes */
	uint16_t creator1;  /* reserved. */
	uint16_t creator2;  /* reserved. */
	uint32_t offset;    /* the offset, i.e. starting address,
						of the byte where the bitmap data can be found. */
} bmp_header_t;

typedef struct {
	uint32_t header_sz;     /* the size of this header (40 bytes) */
	uint32_t width;         /* the bitmap width in pixels */
	uint32_t height;        /* the bitmap height in pixels */
	uint16_t nplanes;       /* the number of color planes being used.
							Must be set to 1. */
	uint16_t depth;         /* the number of bits per pixel,
							which is the color depth of the image.
							Typical values are 1, 4, 8, 16, 24 and 32. */
	uint32_t compress_type; /* the compression method being used.
							See also bmp_compression_method_t. */
	uint32_t bmp_bytesz;    /* the image size. This is the size of the raw bitmap
							data (see below), and should not be confused
							with the file size. */
	uint32_t hres;          /* the horizontal resolution of the image.
							(pixel per meter) */
	uint32_t vres;          /* the vertical resolution of the image.
							(pixel per meter) */
	uint32_t ncolors;       /* the number of colors in the color palette,
							or 0 to default to 2<sup><i>n</i></sup>. */
	uint32_t nimpcolors;    /* the number of important colors used,
							or 0 when every color is important;
							generally ignored. */
} bmp_dib_v3_header_t;

__global__ void global_defaultgaussfilter(
	unsigned int *input,
	unsigned int *output,
	double *matrix,	int Nh,
	int width,int height,
	int ddxx, int ddyy)
{
	assert(ddxx*ddyy==0);

	if(ddxx>0) for (int id = blockDim.x*blockIdx.x + threadIdx.x;
	id < width*height;
	id += blockDim.x*gridDim.x) {
		int x = id % width;
		int y = id / width;

		double s1 = 0;
		double s2 = 0;
		for (int dx = -Nh; dx <= Nh; dx++) {
			if (x + dx < 0) continue;
			if (x + dx >= width) continue;
			int value = (double)(unsigned int)input[id + dx];
			s1 += value*matrix[Nh + dx];
			s2 += matrix[Nh + dx];
		}
		output[id] = (unsigned int)(s1 / s2);
	}
	if(ddyy>0) for (int id = blockDim.x*blockIdx.x + threadIdx.x;
	id < width*height;
	id += blockDim.x*gridDim.x) {
		int x = id % width;
		int y = id / width;

		double s1 = 0;
		double s2 = 0;
		for (int dy = -Nh; dy <= Nh; dy++) {
			if (y + dy < 0) continue;
			if (y + dy >= height) continue;
			int value = (double)(unsigned int)input[id + width*dy];
			s1 += value*matrix[Nh + dy];
			s2 += matrix[Nh + dy];
		}
		output[id] = (unsigned int)(s1 / s2);
	}
}


__host__ void host_defaultgaussfilter(
	int gridSize, int blockSize, 
	unsigned int *r, unsigned int *g, unsigned int *b, 
	double *matrix, int Nh, 
	int width, int height)
{
	unsigned int *channel[3] = { r,g,b };

	// channel - массив данных
	// sigma - коэффициент размытия
	// width - ширина
	// height - высота

	hipError_t err;
	unsigned int *device_a;
	unsigned int *device_b;
	double *device_matrix;

	err = hipMalloc((void**)&device_a, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_b, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_matrix, (2 * Nh + 1) * sizeof(double));
	hipMemcpy(device_matrix, matrix, (2 * Nh + 1)*sizeof(double), hipMemcpyHostToDevice);

	int blocks = (gridSize > 0)? gridSize : min(15, (int)sqrt(width*height));
	int threads = (blockSize > 0)? blockSize : min(15, (int)sqrt(width*height));

	for(int j=0; j<3; j++) {
		hipMemcpy(device_a, channel[j], width*height*sizeof(unsigned int), hipMemcpyHostToDevice);

		global_defaultgaussfilter <<< blocks, threads >>>(device_a, device_b, device_matrix, Nh, width, height, 0, 1);
		global_defaultgaussfilter <<< blocks, threads >>>(device_b, device_a, device_matrix, Nh, width, height, 1, 0);

		hipMemcpy(channel[j], device_a, width*height*sizeof(unsigned int), hipMemcpyDeviceToHost);
	}

	// Освобождаем память на устройстве
	hipFree((void*)device_a);
	hipFree((void*)device_b);
	hipFree((void*)device_matrix);

	err = err;
}

__global__ void global_recursivegaussfilter(
	unsigned int *input,
	unsigned int *output,
	double *bb,	int p, int q,
	int width,int height,
	int ddxx, int ddyy)
{
	assert(p==1);
	assert(q==3);
	assert(ddxx*ddyy==0);

	// Копируем в локальные переменные для ускорения доступа к значениям
	double b1 = bb[1]/bb[0];
	double b2 = bb[2]/bb[0];
	double b3 = bb[3]/bb[0];
	double B = 1.0 - (b1+b2+b3);

	if(ddxx>0) for (int y = blockDim.x*blockIdx.x + threadIdx.x;
	y < height;
	y += blockDim.x*gridDim.x) {
		int id = y*width;
		if(width>1) output[id]=B*input[id];id++;
		if(width>2) output[id]=B*input[id]+b1*output[id-1];id++;
		if(width>3) output[id]=B*input[id]+b1*output[id-1]+b2*output[id-2];id++;
		for(; id<width; id++) 
			output[id]=B*input[id]+b1*output[id-1]+b2*output[id-2]+b3*output[id-3];
	}
	if(ddxx<0) for (int y = blockDim.x*blockIdx.x + threadIdx.x;
	y < height;
	y += blockDim.x*gridDim.x) {
		int id = y*width+width;
		id--;if(width>1) output[id]=B*input[id];
		id--;if(width>2) output[id]=B*input[id]+b1*output[id+1];
		id--;if(width>3) output[id]=B*input[id]+b1*output[id+1]+b2*output[id+2];
		for(id--; id>=0; id--) 
			output[id]=B*input[id]+b1*output[id+1]+b2*output[id+2]+b3*output[id+3];
	}
	if(ddyy>0) for (int x = blockDim.x*blockIdx.x + threadIdx.x;
	x < width;
	x += blockDim.x*gridDim.x) {
		int id = x;
		if(height>1) output[id]=B*input[id];id+=width;
		if(height>2) output[id]=B*input[id]+b1*output[id-width];id+=width;
		if(height>3) output[id]=B*input[id]+b1*output[id-width]+b2*output[id-2*width];id+=width;
		for(int y=3; y<height; y++, id+=width) 
			output[id]=B*input[id]+b1*output[id-width]+b2*output[id-2*width]+b3*output[id-3*width];
	}
	if(ddyy<0) for (int x = blockDim.x*blockIdx.x + threadIdx.x;
	x < width;
	x += blockDim.x*gridDim.x){
		int id = height*width+x;
		id-=width;if(height>1) output[id]=B*input[id];
		id-=width;if(height>2) output[id]=B*input[id]+b1*output[id+width];
		id-=width;if(height>3) output[id]=B*input[id]+b1*output[id+width]+b2*output[id+2*width];
		for(id-=width; id>=0; id-=width) 
			output[id]=B*input[id]+b1*output[id+width]+b2*output[id+2*width]+b3*output[id+3*width];
	}
}

__host__ void host_recursivegaussfilter(
	int gridSize, int blockSize, 
	unsigned int *r, unsigned int *g, unsigned int *b, 
	double *bb, int p, int q, 
	int width, int height)
{

	unsigned int *channel[3] = { r,g,b };

	// channel - массив данных
	// sigma - коэффициенты
	// width - ширина
	// height - высота

	hipError_t err;
	unsigned int *device_a;
	unsigned int *device_b;
	double *device_bb;

	err = hipMalloc((void**)&device_a, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_b, width*height*sizeof(unsigned int));
	err = hipMalloc((void**)&device_bb, (p+q) * sizeof(double));
	hipMemcpy(device_bb, bb, (p+q)*sizeof(double), hipMemcpyHostToDevice);

	int blocks = (gridSize > 0)? gridSize : min(15, (int)sqrt(max(width,height)));
	int threads = (blockSize > 0)? blockSize : min(15, (int)sqrt(max(width,height)));

	for(int j=0; j<3; j++) {
		hipMemcpy(device_a, channel[j], width*height*sizeof(unsigned int), hipMemcpyHostToDevice);

		global_recursivegaussfilter <<< blocks, threads >>>(device_a, device_b, device_bb, p, q, width, height, 0, 1);
		global_recursivegaussfilter <<< blocks, threads >>>(device_b, device_a, device_bb, p, q, width, height, 0, -1);
		global_recursivegaussfilter <<< blocks, threads >>>(device_a, device_b, device_bb, p, q, width, height, 1, 0);
		global_recursivegaussfilter <<< blocks, threads >>>(device_b, device_a, device_bb, p, q, width, height, -1, 0);

		hipMemcpy(channel[j], device_a, width*height*sizeof(unsigned int), hipMemcpyDeviceToHost);
	}

	// Освобождаем память на устройстве
	hipFree((void*)device_a);
	hipFree((void*)device_b);
	hipFree((void*)device_bb);

	err = err;
}

#define RECURSIVE 1
#define DEFAULT 0

int main(int argc, char* argv[])
{
	int i, j;

	std::cout << title << std::endl;

	// Find/set the device.
	int device_count = 0;
	hipGetDeviceCount(&device_count);
	for (i = 0; i < device_count; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		std::cout << "Running on GPU " << i << " (" << properties.name << ")" << std::endl;
	}

	if (argc < 5){
		printf("Usage :\t%s filter sigma inputfilename.bmp ouputfilename.bmp\n", argv[0]);
		printf("filter :\tgauss recursivegauss\n");
		exit(-1);
	}

	// Получаем параметры - имена файлов

	char *filter = argv[1];
	double sigma = atof(argv[2]);
	char *inputFileName = argv[3];
	char *outputFileName = argv[4];
	int gridSize = (argc>5)?atoi(argv[5]):0;
	int blockSize = (argc>6)?atoi(argv[6]):0;
	int mode =(strcmp(filter,"gauss")==0)?DEFAULT:RECURSIVE;

	printf("Title :\t%s\n", title);
	printf("Description :\t%s\n", description);
	printf("Filter :\t%s\n", filter);
	printf("Sigma :\t%le\n", sigma);
	printf("Input file name :\t%s\n", inputFileName);
	printf("Output file name :\t%s\n", outputFileName);

	FILE *file = fopen(inputFileName, "rb");
	if (!file) { 
		fprintf(stderr, "Open file error (%s)\n", inputFileName); 
		fflush(stderr); 
		exit(-1); 
	}
	fseek(file, 0L, SEEK_END);
	long size = ftell(file);
	unsigned char *buffer = (unsigned char *)malloc((size_t)size);
	fseek(file, 0L, SEEK_SET);
	fread((void*)buffer, (size_t)1, (size_t)size, file);
	fclose(file);

	uint32_t width = *(uint32_t *)&buffer[BMPWIDTH];
	uint32_t height = *(uint32_t *)&buffer[BMPHEIGHT];
	uint32_t file_size = *(uint32_t *)&buffer[BMPFILESIZE];
	uint32_t offset = *(uint32_t *)&buffer[BMPOFFSET];
	uint16_t bits_per_pixel = *(uint16_t *)&buffer[BMPBITSPERPIXEL];
	uint16_t bytes_per_pixel = ((int)((bits_per_pixel+7)/8));
	uint32_t bytes_per_line = ((int)((bits_per_pixel * width+31)/32))*4; // http://en.wikipedia.org/wiki/BMP_file_format

	printf("BMP image size :\t%ld x %ld\n", width, height);
	printf("BMP file size :\t%ld\n", file_size);
	printf("BMP pixels offset :\t%ld\n", offset);
	printf("BMP bits per pixel :\t%d\n", bits_per_pixel);
	printf("BMP bytes per pixel :\t%d\n", bytes_per_pixel);
	printf("BMP bytes per line :\t%d\n", bytes_per_line);

	uint8_t *pixels =(uint8_t *)&buffer[offset];
	uint8_t *pixel;
	uint32_t x, y;

	/* выделение памяти под байтовые цветовые каналы */
	int count = width*height;
	unsigned int* channel[3];
	for (j = 0; j < 3; j++)
		channel[j] = (unsigned int*)malloc(count*sizeof(unsigned int));

	int pos = 0;
	for (y = 0; y < height; y++) {
		for (x = 0; x < width; x++) {
			pixel = &pixels[y*bytes_per_line+x*bytes_per_pixel];
			channel[0][pos] = (unsigned int)pixel[0];
			channel[1][pos] = (unsigned int)pixel[1];
			channel[2][pos] = (unsigned int)pixel[2];
			pos++;
		}
	}

	switch(mode){
	case RECURSIVE:{
		int p = 1, q = 3;
		double * bb = (double*)malloc((p+q)*sizeof(double));

		//// http://mognovse.ru/qgi-chekan-rostislav-vladimirovich.html
		//// cтандартные рекомендации по выбору константы q представляют собой:
		//double qq;
		//if(sigma>2.5) qq=0.98711*sigma - 0.96330;
		//else if(sigma>0.5) qq=3.97156 - 4.14554*sqrt(1.0 - 0.26891*sigma);
		//else qq=3.97156 - 4.14554*sqrt(1.0 - 0.26891*sigma);

		//double qq2 = qq*qq;
		//double qq3 = qq2*qq;

		//bb[0] = 1.5725 + 2.44413*qq + 1.4281*qq2;
		//bb[1] = 2.44413*qq + 2.85619*qq2 + 1.26661*qq3;
		//bb[2] = -1.4281*qq2 - 1.26661*qq3;
		//bb[3] = 0.422205*qq3;

		// http://habrahabr.ru/post/151157/
		double sigma_inv_4;

		sigma_inv_4 = sigma*sigma; sigma_inv_4 = 1.0/(sigma_inv_4*sigma_inv_4);

		double coef_A = sigma_inv_4*(sigma*(sigma*(sigma*1.1442707+0.0130625)-0.7500910)+0.2546730);
		double coef_W = sigma_inv_4*(sigma*(sigma*(sigma*1.3642870+0.0088755)-0.3255340)+0.3016210);
		double coef_B = sigma_inv_4*(sigma*(sigma*(sigma*1.2397166-0.0001644)-0.6363580)-0.0536068);

		double z0_abs   = exp(coef_A);

		double z0_real  = z0_abs*cos(coef_W);
		double z0_im    = z0_abs*sin(coef_W);
		double z2       = exp(coef_B);

		double z0_abs_2 = z0_abs*z0_abs;

		bb[3] = 1.0/(z2*z0_abs_2);
		bb[2] = -(2*z0_real+z2)*bb[3];
		bb[1] = (z0_abs_2+2*z0_real*z2)*bb[3];
		bb[0] = 1.0;

		clock_t t1, t2;
		t1 = clock();

		host_recursivegaussfilter(gridSize, blockSize, 
			channel[0], channel[1], channel[2], 
			bb, p, q,
			width, height);

		t2 = clock();
		double seconds = ((double)(t2-t1))/CLOCKS_PER_SEC;
		printf("Execution time :\t%le\n", seconds);

		free(bb);
				   }
				   break;
	default:{
		// Nh - полуразмер
		int Nh = (int)(3 * sigma);

		double * matrix = (double*)malloc((2 * Nh + 1)*sizeof(double));
		for (int i = -Nh; i <= Nh; i++) matrix[Nh + i] = exp(-(double)i*i / (2.0*sigma*sigma));

		clock_t t1, t2;
		t1 = clock();

		host_defaultgaussfilter(gridSize, blockSize, 
			channel[0], channel[1], channel[2], 
			matrix, Nh, 
			width, height);

		t2 = clock();
		double seconds = ((double)(t2-t1))/CLOCKS_PER_SEC;
		printf("Execution time :\t%le\n", seconds);

		free(matrix);
			}
			break;
	}

	pos = 0;
	for (y = 0; y < height; y++) {
		for (x = 0; x < width; x++) {
			pixel = &pixels[y*bytes_per_line+x*bytes_per_pixel];
			pixel[0] = (uint8_t)channel[0][pos];
			pixel[1] = (uint8_t)channel[1][pos];
			pixel[2] = (uint8_t)channel[2][pos];
			pos++;
		}
	}
	/* высвобождаем массивы */
	for (j = 0; j < 3; j++)
		free(channel[j]);

	/* выводим результаты */
	file = fopen(outputFileName, "wb");
	if (!file) {
		fprintf(stderr, "Open file error (%s)\n", outputFileName); 
		fflush(stderr);
		exit(-1);
	}
	fwrite((void*)buffer, (size_t)1, (size_t)size, file);
	printf("Output file size :\t%ld\n", size);
	free(buffer);
	fclose(file);

	hipDeviceReset();

	exit(0);
}